#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>


__global__ void vecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void vecAddCPU(float* A, float* B, float* C, int N){
  for(int i=0;i<N;i++){
    C[i] = A[i] + B[i];
  }
}

int main(){
    int N = 10000000;
    size_t size = N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    float* h_C_CPU = (float*)malloc(size);


    for(int i=0;i<N;i++){
      h_A[i] = i;
      h_B[i] = i;
    }

    //------------CPU Execution--------------

    auto start_cpu = std::chrono::high_resolution_clock::now();
    vecAddCPU(h_A, h_B, h_C_CPU, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_time = end_cpu - start_cpu;


    //--------------GPU Execution------------

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    //cuda event intialization
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //-------------Results------------
    for(int i=0;i<N;i++){
      if(h_C[i]!=h_C_CPU[i]) std::cout<<"incorrect"<<std::endl;
    }
    std::cout << "CPU time: " << cpu_time.count() << " ms\n";
    std::cout << "GPU time: " << milliseconds << " ms\n";
    return 0;
}
